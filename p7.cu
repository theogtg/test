/****************************************************************************************************
 * Tyler Griffith                                                                                   *
 * October 25th, 2018                                                                               *
 * Project 7: Matrix Mult on GPU                                                                    *
 * CSC-4310-01 PROF: R. Shore                                                                       *
 * Desc: Use one thread to compute each element of the solution                                     *
         matrix                                                                                     *
 * To Compile: nvcc p6.cu -o cuda_mult_v1                                                           *
 * To Run: ./cuda_mult_v1 <device #> <tile width> <matrix A file> <matrix B file> <matrix A*B file> *
 ****************************************************************************************************/
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

int getN(char* fileName);
int* readMatrix(char* fileName);
void writeMatrix(int *x, int n, char* fileName);

__global__ 
void MatrixMultKernel(int *a, int *b, int *ab, int size){
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int prod = 0;
        if(col < size && row < size){
                for(int i=0; i<size; i++){
                        prod += a[row*size+i] * b[i*size+col];
                }
                ab[row*size+col] = prod;
        }
}

 int main(int argc, char *argv[]){
   //reading in the matrix
   int n;

   //make sure correct syntax is used
   if(argc != 6){
      printf("Error! You do not have 5 elements to your command!\n");
      printf("To multiply 2 matricies please use the following syntax:\n");
      printf("./cuda_mult_v1 <device #> <tile width> <matrix A file> <matrix B file> <matrix A*B file>\n");
      exit(1);
   }

   //variable declaration
   long dNum = strtol(argv[1], NULL, 10);
   int deviceNum = int(dNum);
   long width = strtol(argv[2], NULL, 10);
   const int tileWidth = (int)width;
   int *matrixA, *matrixB, *matrixC, *d_a, *d_b, *d_c;

   //set device
   hipSetDevice(deviceNum);
   
   //get n
   n = getN(argv[3]);

   //file I/O
   matrixA = readMatrix(argv[3]);
   matrixB = readMatrix(argv[4]);

   //allocate and initialize result
   matrixC = new int[n*n];
   for (int i = 0; i < n*n; ++i) {
      matrixC[i] = 0;
   }

   //cuda timing
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   int size = n*n*sizeof(int);
   //allocate cuda memory and copy to global memory
   hipMalloc((void **)&d_a, size);
   hipMemcpy(d_a, matrixA, size, hipMemcpyHostToDevice);
   hipMalloc((void **)&d_b, size);
   hipMemcpy(d_b, matrixB, size, hipMemcpyHostToDevice);
   
   //allocate memory for result
   hipMalloc((void **)&d_c, size);
   hipMemset(d_c, 0, size);

   dim3 dimGrid(tileWidth, tileWidth);
   dim3 dimBlock(tileWidth>>1, tileWidth>>1);


   //start cuda timing
   hipEventRecord(start);
   //kernel call
   MatrixMultKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
   //end cuda timing
   hipEventRecord(stop);

   //copy answer back to CPU
   hipMemcpy(matrixC, d_c, n*n*sizeof(int), hipMemcpyDeviceToHost);
   
   //stop timing
   hipEventSynchronize(stop);
   float ms = 0;
   hipEventElapsedTime(&ms, start, stop);
   //check for error
   hipError_t error = hipGetLastError();
   if(error != hipSuccess){
      //print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
   }

   //print timing
   printf("Computation completed in %fms", ms);
   writeMatrix(matrixC, n, argv[5]);

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
   
   return 0;
}

int getN(char* fileName){
    int n;
    FILE *inFile;
    inFile = fopen(fileName, "r");

   fscanf(inFile, "%d", &n);
   fclose(inFile);
   return n;
}

int* readMatrix(char* fileName){
   int n;
   FILE *inFile;
   inFile = fopen(fileName, "r");

   fscanf(inFile, "%d", &n);

   //allocate memory
   int *x = (int*)malloc(n*n*sizeof(int));

   //read in matrix
   for(int row=0; row<n; row++){
      for(int col=0; col<n; col++){
         fscanf(inFile, "%d", &x[row*n+col]);
      }
   }
   fclose(inFile);
   return x;
}

void writeMatrix(int *x, int n, char* fileName){
   FILE *outFile;
   outFile = fopen(fileName, "w");
   for(int row=0; row<n; row++){
      for(int col=0; col<n; col++)
         fprintf(outFile, "%d ", x[row*n+col]);
      fprintf(outFile, "\n");
   }
   fclose(outFile);
}